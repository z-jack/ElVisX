#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
// The MIT License
//
// Copyright (c) 2006 Scientific Computing and Imaging Institute,
// University of Utah (USA)
//
// License for the specific language governing rights and limitations under
// Permission is hereby granted, free of charge, to any person obtaining a
// copy of this software and associated documentation files (the "Software"),
// to deal in the Software without restriction, including without limitation
// the rights to use, copy, modify, merge, publish, distribute, sublicense,
// and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included
// in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
// OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
// THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
// DEALINGS IN THE SOFTWARE.
//
///////////////////////////////////////////////////////////////////////////////

#ifndef ELVIS_EXTENSIONS_JACOB_EXTENSION_OPTIX_PRISM_CU
#define ELVIS_EXTENSIONS_JACOB_EXTENSION_OPTIX_PRISM_CU

#include <ElVis/Core/matrix.cu>
#include <optix_cuda.h>
#include <optix_math.h>
#include <optixu/optixu_aabb.h>
#include <ElVis/Core/matrix.cu>
#include <ElVis/Core/CutSurfacePayloads.cu>
#include <ElVis/Core/VolumeRenderingPayload.cu>
#include <ElVis/Core/typedefs.cu>
#include <ElVis/Core/jacobi.cu>
#include <ElVis/Core/util.cu>
#include <ElVis/Core/OptixVariables.cu>
#include <ElVis/Core/Interval.hpp>
#include <ElVis/Core/IntervalPoint.cu>
#include <ElVis/Extensions/JacobiExtension/PrismCommon.cu>

// The vertices associated with this prism.
// Prism has 6 vertices.
rtBuffer<ElVisFloat4> PrismVertexBuffer;

// The vertices associated with each face.
// Faces 0-2 are quads and all four elements are used.
// Faces 3 and 4 are triangles
rtBuffer<uint4> Prismvertex_face_index;

// The planes associated with each face.
rtBuffer<ElVisFloat4> PrismPlaneBuffer;

// The coefficients to evaluate the scalar field.
rtBuffer<ElVisFloat> PrismCoefficients;
rtBuffer<uint> PrismCoefficientIndices;

rtBuffer<uint3> PrismDegrees;

rtDeclareVariable(int, intersectedPrismId, attribute IntersectedHex, );


#endif
