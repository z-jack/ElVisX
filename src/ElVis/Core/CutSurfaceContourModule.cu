#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
// The MIT License
//
// Copyright (c) 2006 Scientific Computing and Imaging Institute,
// University of Utah (USA)
//
// License for the specific language governing rights and limitations under
// Permission is hereby granted, free of charge, to any person obtaining a
// copy of this software and associated documentation files (the "Software"),
// to deal in the Software without restriction, including without limitation
// the rights to use, copy, modify, merge, publish, distribute, sublicense,
// and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included
// in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
// OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
// THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
// DEALINGS IN THE SOFTWARE.
//
///////////////////////////////////////////////////////////////////////////////

#ifndef ELVIS_CUT_SURFACE_CONTOUR_MODULE_CU
#define ELVIS_CUT_SURFACE_CONTOUR_MODULE_CU

#include <optix_cuda.h>
#include <optix_math.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_aabb.h>
#include <ElVis/Core/CutSurfacePayloads.cu>
#include <ElVis/Core/ConvertToColor.cu>
#include <ElVis/Core/Float.h>
#include <ElVis/Core/Interval.hpp>
#include <ElVis/Core/ElementId.h>

rtBuffer<ElVisFloat, 2> ContourSampleBuffer;
//rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<ElVisFloat3, 2> ReferencePointAtIntersectionBuffer;
rtBuffer<unsigned int, 2> ElementIdAtIntersectionBuffer;
rtBuffer<unsigned int, 2> ElementTypeAtIntersectionBuffer;

rtDeclareVariable(rtObject, CutSurfaceContourGeometry, , );

rtBuffer<ElVisFloat, 1> Isovalues;

rtDeclareVariable(int, TreatElementBoundariesAsDiscontinuous, , );

ELVIS_DEVICE void GetPixelOffset(int curx, int cury, int offsetx, int offsety, int& id, int& type)
{
    int2 newIndex = make_int2(curx + offsetx, cury + offsety);

    if( newIndex.x >= 0 && newIndex.x < color_buffer.size().x &&
        newIndex.y >= 0 && newIndex.y < color_buffer.size().y )
    {
        uint2 index = make_uint2(newIndex.x, newIndex.y);
        ELVIS_PRINTF("testing pixel (%d, %d) \n",
                     newIndex.x, newIndex.y);
        id = ElementIdBuffer[index];
        type = ElementTypeBuffer[index];
    }
}

RT_PROGRAM void CutSurfaceMeshProgram()
{
    bool isCrossing = false;

    int curPixelId = ElementIdBuffer[launch_index];
    int curPixelType = ElementTypeBuffer[launch_index];

    for(int i = -1; i <= 1; ++i)
    {
        for(int j = -1; j <= 1; ++j)
        {
            int id = curPixelId;
            int type = curPixelType;

            GetPixelOffset(launch_index.x, launch_index.y, i, j, id, type);
            ELVIS_PRINTF("Pixel (%d, %d) has (%d, %d) and adjacent (%d,%d) has (%d, %d)\n",
                         launch_index.x, launch_index.y, curPixelId, curPixelType,
                         i, j, id, type);
            if( id >= 0 && type >= 0 )
            {
                isCrossing |= ((id != curPixelId) || (type != curPixelType));
            }

            if( isCrossing ) break;
        }
    }


    if( isCrossing )
    {
        raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
        normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
    }

}

// raw_color_buffer 
RT_PROGRAM void SamplePixelCornersRayGenerator()
{
    ELVIS_PRINTF("SamplePixelCornersRayGenerator\n");
    // Note - there are occlusion issues here.
    ElVisFloat2 screen = MakeFloat2(color_buffer.size());
    ElVisFloat2 pixelSize = MAKE_FLOAT(2.0)/screen;
    
    ElVisFloat x = MAKE_FLOAT(-1.0);
    ElVisFloat y = MAKE_FLOAT(-1.0);
    ElVisFloat2 pixelOffset = MakeFloat2(x, y)/MAKE_FLOAT(2.0);
    
    ElVisFloat2 d = MakeFloat2(launch_index) / screen * MAKE_FLOAT(2.0) - MAKE_FLOAT(1.0);
    d = d + pixelSize * pixelOffset;
    
    ElVisFloat3 ray_origin = eye;
    ElVisFloat3 ray_direction = normalize(d.x*U + d.y*V + W);

    optix::Ray ray = optix::make_Ray(ConvertToFloat3(ray_origin), ConvertToFloat3(ray_direction), 1, 1e-3f, RT_DEFAULT_MAX);
    CutSurfaceScalarValuePayload payload;
    
    payload.Initialize();
    payload.isValid = false;
    payload.scalarValue = ELVIS_FLOAT_MAX;
    payload.Normal = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    payload.Color = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
    rtTrace(SurfaceGeometryGroup, ray, payload);
    //rtTrace(CutSurfaceContourGeometry, ray, payload);
    
    ContourSampleBuffer[launch_index] = payload.scalarValue;
}

RT_PROGRAM void ContourMiss()
{
}

RT_PROGRAM void MarkContourPixels()
{
    ELVIS_PRINTF("MarkContourPixels\n");
    // Corner testing.
    uint2 c0_index = make_uint2(launch_index.x, launch_index.y);
    uint2 c1_index = make_uint2(launch_index.x, launch_index.y);;
    c1_index.x += 1;
    uint2 c2_index = make_uint2(launch_index.x, launch_index.y);;
    c2_index.y += 1;
    uint2 c3_index = make_uint2(launch_index.x, launch_index.y);;
    c3_index.x += 1;
    c3_index.y += 1;
    
    ElVisFloat c0 = ContourSampleBuffer[c0_index];
    ElVisFloat c1 = ContourSampleBuffer[c1_index];
    ElVisFloat c2 = ContourSampleBuffer[c2_index];
    ElVisFloat c3 = ContourSampleBuffer[c3_index];
    
    // The 5000 are to get around a bug for the demos but needs to be fixed.
    bool allSamplesValid = true;
    allSamplesValid = (c0 != ELVIS_FLOAT_MAX) &&
        (c1 != ELVIS_FLOAT_MAX) &&
        (c2 != ELVIS_FLOAT_MAX) &&
        (c3 != ELVIS_FLOAT_MAX) &&
        c0 < 5000 && c1 < 5000 && c2 < 5000 && c3 < 5000;
            
    if( !allSamplesValid ) return;
    
    ELVIS_PRINTF("MarkContourPixels: All corners have a sample\n");
    for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
    {
        ElVisFloat isovalue = Isovalues[isoValueIndex];
        bool lowerThanOneValue = (isovalue <= c0) || (isovalue <= c1) || (isovalue <= c2) || (isovalue <= c3);
        bool higherThanOneValue = (isovalue >= c0) || (isovalue >= c1) || (isovalue >= c2) || (isovalue >= c3);
        
        if( lowerThanOneValue && higherThanOneValue )
        {
            ELVIS_PRINTF("(%d, %d), Isovalue %f and corners %f, %f, %f, %f\n", launch_index.x, launch_index.y, isovalue, c0, c1, c2, c3);
            raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
    }
}




RT_PROGRAM void MarkContourPixelsWithSutrms()
{
    // Corner testing.
    uint2 c0_index = make_uint2(launch_index.x, launch_index.y);
    uint2 c1_index = make_uint2(launch_index.x, launch_index.y);;
    c1_index.x += 1;
    uint2 c2_index = make_uint2(launch_index.x, launch_index.y);;
    c2_index.y += 1;
    uint2 c3_index = make_uint2(launch_index.x, launch_index.y);;
    c3_index.x += 1;
    c3_index.y += 1;
    
    ElVisFloat c0 = ContourSampleBuffer[c0_index];
    ElVisFloat c1 = ContourSampleBuffer[c1_index];
    ElVisFloat c2 = ContourSampleBuffer[c2_index];
    ElVisFloat c3 = ContourSampleBuffer[c3_index];
    
    // The 5000 are to get around a bug for the demos but needs to be fixed.
    bool allSamplesValid = true;
    allSamplesValid = (c0 != ELVIS_FLOAT_MAX) &&
        (c1 != ELVIS_FLOAT_MAX) &&
        (c2 != ELVIS_FLOAT_MAX) &&
        (c3 != ELVIS_FLOAT_MAX) &&
        c0 < 5000 && c1 < 5000 && c2 < 5000 && c3 < 5000;
            
    if( !allSamplesValid ) return;
    
    bool found = false;
    for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
    {       
        ElVisFloat isovalue = Isovalues[isoValueIndex];
        bool lowerThanOneValue = (isovalue <= c0) || (isovalue <= c1) || (isovalue <= c2) || (isovalue <= c3);
        bool higherThanOneValue = (isovalue >= c0) || (isovalue >= c1) || (isovalue >= c2) || (isovalue >= c3);
        
        if( lowerThanOneValue && higherThanOneValue )
        {
            found = true;
            //ELVIS_PRINTF("(%d, %d), Isovalue %f and corners %f, %f, %f, %f\n", launch_index.x, launch_index.y, isovalue, c0, c1, c2, c3);
            raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
    }

    if( found ) return;

    // At this point all corners were either too low or too high, but we don't know for sure if that means there is no crossing.
    // Options:
    // Use interval analysis along an edge to see if it is even possible.  This approach can be useful to reject regions that 
    // cannot have the contour, but will be unable to provide tight bounds.  The best this approach could do is provide tight bounds 
    // along a rectangle in reference space, but not along a curve.  So the bounds could indicate that the isovalue occurs inside the 
    // rectangle when it doesn't touch the curve, which would be a false positive.

    // The next option is to sample along each edge and project onto a n degree polynomial, which can then be used to, via Sturm's method, 
    // to find how many real roots lie along the edge.  This is the option implemented here.

    // PROBLEM - the projection will fail at element boundaries (c0 and all that).  So at element boundaries we can't do intervals, we can't do 
    // polynomial projection.  What can we do?  We can say that this method works on pixels that span a single element/surface, and other techniques
    // are used for boundary cases.  Maybe look at edge calculations from adjacent pixels

    // Are there numerical integration techniques that handle discontinuities in the derivative well?  If so, can the integral of a function tell 
    // us anything about if an isovalue exists.

    // Order of the approximating polynomial, which should be obtained from the elements that are being sampled, or user specified.

}

__device__ __forceinline__ ElVisFloat3 CalculateRayDirection(const uint2& pixelIndex, const ElVisFloat2& offset)
{
    ElVisFloat2 screen = MakeFloat2(color_buffer.size());
    ElVisFloat2 pixelSize = MAKE_FLOAT(2.0)/screen;
    
    ElVisFloat2 pixelOffset = offset/MAKE_FLOAT(2.0);
    
    ElVisFloat2 d = MakeFloat2(launch_index) / screen * MAKE_FLOAT(2.0) - MAKE_FLOAT(1.0);
    d = d + pixelSize * pixelOffset;
    
    //float3 ray_origin = eye;
    ElVisFloat3 ray_direction = normalize(d.x*U + d.y*V + W);
    return ray_direction;
}

// raw_color_buffer 
RT_PROGRAM void SamplePixelCornersRayGeneratorForCategorization()
{
    ELVIS_PRINTF("SamplePixelCornersRayGenerator\n");
    // Note - there are occlusion issues here.
    ElVisFloat2 screen = MakeFloat2(color_buffer.size());
    ElVisFloat2 pixelSize = MAKE_FLOAT(2.0)/screen;
    
    ElVisFloat x = MAKE_FLOAT(-1.0);
    ElVisFloat y = MAKE_FLOAT(-1.0);
    ElVisFloat2 pixelOffset = MakeFloat2(x, y)*MAKE_FLOAT(.5);
    
    ElVisFloat2 d = MakeFloat2(launch_index) / screen * MAKE_FLOAT(2.0) - MAKE_FLOAT(1.0);
    d = d + pixelSize * pixelOffset;
    
    ElVisFloat3 ray_origin = eye;
    ElVisFloat3 ray_direction = CalculateRayDirection(launch_index, MakeFloat2(MAKE_FLOAT(-1.0), MAKE_FLOAT(-1.0)));//normalize(d.x*U + d.y*V + W);

    optix::Ray ray = optix::make_Ray(ConvertToFloat3(ray_origin), ConvertToFloat3(ray_direction), 0, 1e-3f, RT_DEFAULT_MAX);
    CutSurfaceScalarValuePayload payload;
    
    payload.Initialize();
    payload.isValid = false;
    payload.scalarValue = ELVIS_FLOAT_MAX;
    payload.Normal = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    payload.Color = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
    rtTrace(SurfaceGeometryGroup, ray, payload);
    
    ContourSampleBuffer[launch_index] = payload.scalarValue;
    ReferencePointAtIntersectionBuffer[launch_index] = payload.ReferenceIntersectionPoint;
    ElementIdAtIntersectionBuffer[launch_index] = payload.elementId;
    ElementTypeAtIntersectionBuffer[launch_index] = payload.elementType;

}

__device__ __forceinline__ ElVis::Interval<ElVisFloat> EvaluatePrismBetweenReferencePoints(unsigned int elementId, const ElVisFloat3& p0, const ElVisFloat3& p1)
{
    // ElVis::Interval<ElVisFloat> r0(fminf(p0.x, p1.x), fmaxf(p0.x, p1.x));
    // ElVis::Interval<ElVisFloat> s0(fminf(p0.y, p1.y), fmaxf(p0.y, p1.y));
    // ElVis::Interval<ElVisFloat> t0(fminf(p0.z, p1.z), fmaxf(p0.z, p1.z));
    // return EvaluatePrism(elementId, r0, s0, t0);
  return ElVis::Interval<ElVisFloat>();
}

__device__ __forceinline__ ElVis::Interval<ElVisFloat> EvaluatePrismEdge(unsigned int elementId, uint2 i0, uint2 i1)
{
    // ElVisFloat3 p0 = ReferencePointAtIntersectionBuffer[i0];
    // ElVisFloat3 p1 = ReferencePointAtIntersectionBuffer[i1];
    
    // return EvaluatePrismBetweenReferencePoints(elementId, p0, p1);
  return ElVis::Interval<ElVisFloat>();
}

//__device__ __forceinline__ ElVis::Interval<ElVisFloat> SubdivideInterval1(unsigned int elementId,
//                                  const ElVisFloat3& p0, const ElVisFloat3& p1, 
//                                  const ElVisFloat2& cornerOffset0, const ElVisFloat2& cornerOffset1, int numSubdivisions)
//{
//}

// p0 - The reference point at the beginning of the interval.
// p1 - The reference point at the end of the interval.
__device__ __forceinline__ ElVis::Interval<ElVisFloat> SubdivideInterval1(unsigned int elementId,
                                  const ElVisFloat3& p0, const ElVisFloat3& p1, 
                                  const ElVisFloat2& cornerOffset0, const ElVisFloat2& cornerOffset1)
{
    // Input is two pixel corners.  We'll do up to two levels of subdivision to start with and see how that affects
    // the final image.
    // First level is a ray at 1/2 between pixel corners.
    // Seconds level is at 1/4 and 3/4.

    ElVisFloat2 offset;
    if( cornerOffset0.x == cornerOffset1.x )
    {
        // Vertical edge
        offset = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(0.0));
    }
    else
    {
        // horizontal edge.
        offset = MakeFloat2(MAKE_FLOAT(0.0), cornerOffset0.y);
    }

    ElVisFloat3 ray_direction = CalculateRayDirection(launch_index, offset); 

    ElVisFloat3 ray_origin = eye;
    optix::Ray ray = optix::make_Ray(ConvertToFloat3(ray_origin), ConvertToFloat3(ray_direction), 2, 1e-3f, RT_DEFAULT_MAX);
    CutSurfaceScalarValuePayload payload;
    payload.Initialize();
    payload.isValid = false;
    payload.scalarValue = ELVIS_FLOAT_MAX;
    payload.Normal = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    payload.Color = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
    rtTrace(CutSurfaceContourGeometry, ray, payload);   
    
    // Now evaluate the interval math between p0-mid and mid-p1 to see if we can reject
    // this pixel.
    ElVisFloat3 mid = payload.ReferenceIntersectionPoint;
    ElVis::Interval<ElVisFloat> i0 = EvaluatePrismBetweenReferencePoints(elementId, p0, mid);
    ElVis::Interval<ElVisFloat> i1 = EvaluatePrismBetweenReferencePoints(elementId, mid, p1);

    return ElVis::Interval<ElVisFloat>(fminf(i0.GetLow(), i1.GetLow()), fmaxf(i0.GetHigh(), i1.GetHigh()));
}

__device__ __forceinline__ ElVis::Interval<ElVisFloat> SubdivideInterval2(unsigned int elementId,
                                  const ElVisFloat3& p0, const ElVisFloat3& p1, 
                                  const ElVisFloat2& cornerOffset0, const ElVisFloat2& cornerOffset1)
{
    // Input is two pixel corners.  We'll do up to two levels of subdivision to start with and see how that affects
    // the final image.
    // First level is a ray at 1/2 between pixel corners.
    // Seconds level is at 1/4 and 3/4.

    ElVisFloat2 offset[3];
    if( cornerOffset0.x == cornerOffset1.x )
    {
        // Vertical edge
        offset[0] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(-.5));
        offset[1] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(0.0));
        offset[2] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(.5));
    }
    else
    {
        offset[0] = MakeFloat2(MAKE_FLOAT(-.5), cornerOffset0.y);
        offset[1] = MakeFloat2(MAKE_FLOAT(0.0), cornerOffset0.y);
        offset[2] = MakeFloat2(MAKE_FLOAT(.5), cornerOffset0.y);
    }

    ElVisFloat3 ray_direction[] = {
        CalculateRayDirection(launch_index, offset[0]),
        CalculateRayDirection(launch_index, offset[1]),
        CalculateRayDirection(launch_index, offset[2]) }; 

    CutSurfaceScalarValuePayload payload[3];
    ElVisFloat3 ray_origin = eye;

    for(unsigned int i = 0; i < 3; ++i)
    {
        optix::Ray ray = optix::make_Ray(ConvertToFloat3(ray_origin), ConvertToFloat3(ray_direction[i]), 2, 1e-3f, RT_DEFAULT_MAX);

        payload[i].Initialize();
        payload[i].isValid = false;
        payload[i].scalarValue = ELVIS_FLOAT_MAX;
        payload[i].Normal = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        payload[i].Color = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
        rtTrace(CutSurfaceContourGeometry, ray, payload[i]);   
    }

    // Now evaluate the interval math between p0-mid and mid-p1 to see if we can reject
    // this pixel.
    ElVis::Interval<ElVisFloat> i0 = EvaluatePrismBetweenReferencePoints(elementId, p0, payload[0].ReferenceIntersectionPoint);
    ElVis::Interval<ElVisFloat> i1 = EvaluatePrismBetweenReferencePoints(elementId, payload[0].ReferenceIntersectionPoint, payload[1].ReferenceIntersectionPoint);
    ElVis::Interval<ElVisFloat> i2 = EvaluatePrismBetweenReferencePoints(elementId, payload[1].ReferenceIntersectionPoint, payload[2].ReferenceIntersectionPoint);
    ElVis::Interval<ElVisFloat> i3 = EvaluatePrismBetweenReferencePoints(elementId, payload[2].ReferenceIntersectionPoint, p1);

    i0.Combine(i1);
    i0.Combine(i2);
    i0.Combine(i3);
    return i0;
}

__device__ __forceinline__ ElVis::Interval<ElVisFloat> SubdivideInterval3(unsigned int elementId,
                                  const ElVisFloat3& p0, const ElVisFloat3& p1, 
                                  const ElVisFloat2& cornerOffset0, const ElVisFloat2& cornerOffset1)
{
    // Input is two pixel corners.  We'll do up to two levels of subdivision to start with and see how that affects
    // the final image.
    // First level is a ray at 1/2 between pixel corners.
    // Seconds level is at 1/4 and 3/4.

    ElVisFloat2 offset[7];
    if( cornerOffset0.x == cornerOffset1.x )
    {
        // Vertical edge
        offset[0] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(-.75));
        offset[1] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(-.5));
        offset[2] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(-.25));
        offset[3] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(0.0));
        offset[4] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(.25));
        offset[5] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(.5));
        offset[6] = MakeFloat2(cornerOffset0.x, MAKE_FLOAT(.75));
    }
    else
    {
        offset[0] = MakeFloat2(MAKE_FLOAT(-.75), cornerOffset0.y);
        offset[1] = MakeFloat2(MAKE_FLOAT(-.5), cornerOffset0.y);
        offset[2] = MakeFloat2(MAKE_FLOAT(-.25), cornerOffset0.y);
        offset[3] = MakeFloat2(MAKE_FLOAT(0.0), cornerOffset0.y);
        offset[4] = MakeFloat2(MAKE_FLOAT(.25), cornerOffset0.y);
        offset[5] = MakeFloat2(MAKE_FLOAT(.5), cornerOffset0.y);
        offset[6] = MakeFloat2(MAKE_FLOAT(.75), cornerOffset0.y);
    }

    CutSurfaceScalarValuePayload payload[7];
    ElVisFloat3 ray_origin = eye;

    for(unsigned int i = 0; i < 7; ++i)
    {
        ElVisFloat3 ray_direction = CalculateRayDirection(launch_index, offset[i]);
        optix::Ray ray = optix::make_Ray(ConvertToFloat3(ray_origin), ConvertToFloat3(ray_direction), 2, 1e-3f, RT_DEFAULT_MAX);

        payload[i].Initialize();
        payload[i].isValid = false;
        payload[i].scalarValue = ELVIS_FLOAT_MAX;
        payload[i].Normal = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        payload[i].Color = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
        rtTrace(CutSurfaceContourGeometry, ray, payload[i]);   
    }

    // Now evaluate the interval math between p0-mid and mid-p1 to see if we can reject
    // this pixel.
    ElVis::Interval<ElVisFloat> result = EvaluatePrismBetweenReferencePoints(elementId, p0, payload[0].ReferenceIntersectionPoint);
    for(unsigned int i = 0; i < 6; ++i)
    {
        ElVis::Interval<ElVisFloat> i1 = EvaluatePrismBetweenReferencePoints(elementId, payload[i].ReferenceIntersectionPoint, payload[i+1].ReferenceIntersectionPoint);
        result.Combine(i1);
    }
    ElVis::Interval<ElVisFloat> i3 = EvaluatePrismBetweenReferencePoints(elementId, payload[6].ReferenceIntersectionPoint, p1);
    result.Combine(i3);
    return result;
}

RT_PROGRAM void CategorizeMeshPixels()
{
    // Corner testing.
    // c0 = lower left corner
    // c1 = lower right corner
    // c2 = upper left corner
    // c3 = upper right corner
    uint2 c0_index = make_uint2(launch_index.x, launch_index.y);
    uint2 c1_index = make_uint2(launch_index.x, launch_index.y);;
    c1_index.x += 1;
    uint2 c2_index = make_uint2(launch_index.x, launch_index.y);;
    c2_index.y += 1;
    uint2 c3_index = make_uint2(launch_index.x, launch_index.y);;
    c3_index.x += 1;
    c3_index.y += 1;

    ElVis::ElementId id0;
    ElVis::ElementId id1;
    ElVis::ElementId id2;
    ElVis::ElementId id3;




    id0.Id = ElementIdAtIntersectionBuffer[c0_index];
    id1.Id = ElementIdAtIntersectionBuffer[c1_index];
    id2.Id = ElementIdAtIntersectionBuffer[c2_index];
    id3.Id = ElementIdAtIntersectionBuffer[c3_index];

    id0.Type = ElementTypeAtIntersectionBuffer[c0_index];
    id1.Type = ElementTypeAtIntersectionBuffer[c1_index];
    id2.Type = ElementTypeAtIntersectionBuffer[c2_index];
    id3.Type = ElementTypeAtIntersectionBuffer[c3_index];

    if( id0.Id == -1 ||
        id1.Id == -1 ||
        id2.Id == -1 ||
        id3.Id == -1 )
    {
        return;
    }

    bool pixelIsElementBoundary =
     ( id0 != id1 ||
        id1 != id2 ||
        id2 != id3 );

    if( pixelIsElementBoundary )
    {
        raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.5), MAKE_FLOAT(0.5), MAKE_FLOAT(0.5));
        color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
        normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
    }
}

rtDeclareVariable(int, MatchVisual3Contours, , );

RT_PROGRAM void CategorizeContourPixels()
{
    ELVIS_PRINTF("CategorizeContourPixels\n");
    // Corner testing.
    // c0 = lower left corner
    // c1 = lower right corner
    // c2 = upper left corner
    // c3 = upper right corner
    uint2 c0_index = make_uint2(launch_index.x, launch_index.y);
    uint2 c1_index = make_uint2(launch_index.x, launch_index.y);;
    c1_index.x += 1;
    uint2 c2_index = make_uint2(launch_index.x, launch_index.y);;
    c2_index.y += 1;
    uint2 c3_index = make_uint2(launch_index.x, launch_index.y);;
    c3_index.x += 1;
    c3_index.y += 1;
    
    ElVisFloat c0 = ContourSampleBuffer[c0_index];
    ElVisFloat c1 = ContourSampleBuffer[c1_index];
    ElVisFloat c2 = ContourSampleBuffer[c2_index];
    ElVisFloat c3 = ContourSampleBuffer[c3_index];
    
    // The 5000 are to get around a bug for the demos but needs to be fixed.
    bool allSamplesValid = true;
    allSamplesValid = (c0 != ELVIS_FLOAT_MAX) &&
        (c1 != ELVIS_FLOAT_MAX) &&
        (c2 != ELVIS_FLOAT_MAX) &&
        (c3 != ELVIS_FLOAT_MAX) &&
        c0 < 5000 && c1 < 5000 && c2 < 5000 && c3 < 5000;


    ElVisFloat3 visual3BackgroundColor = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));

    ELVIS_PRINTF("CategorizeContourPixels: All Samples Valid %d\n", allSamplesValid);
    if( !allSamplesValid ) 
    {
        if( MatchVisual3Contours )
        {
            raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));;
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
        return;
    }

    ElVis::ElementId id0;
    ElVis::ElementId id1;
    ElVis::ElementId id2;
    ElVis::ElementId id3;


    id0.Id = ElementIdAtIntersectionBuffer[c0_index];
    id1.Id = ElementIdAtIntersectionBuffer[c1_index];
    id2.Id = ElementIdAtIntersectionBuffer[c2_index];
    id3.Id = ElementIdAtIntersectionBuffer[c3_index];

    id0.Type = ElementTypeAtIntersectionBuffer[c0_index];
    id1.Type = ElementTypeAtIntersectionBuffer[c1_index];
    id2.Type = ElementTypeAtIntersectionBuffer[c2_index];
    id3.Type = ElementTypeAtIntersectionBuffer[c3_index];


    bool pixelIsElementBoundary =
     ( id0 != id1 ||
        id1 != id2 ||
        id2 != id3 );

    if( TreatElementBoundariesAsDiscontinuous && pixelIsElementBoundary )
    {
        if( MatchVisual3Contours )
        {
            raw_color_buffer[launch_index] = visual3BackgroundColor;
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
        return;
    }

    ElVisFloat3 elementBoundaryColor = MakeFloat3(MAKE_FLOAT(.5), MAKE_FLOAT(.5), MAKE_FLOAT(.5));
    ElVisFloat3 contourColor = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    ElVisFloat3 ambiguousColor = MakeFloat3(MAKE_FLOAT(.25), MAKE_FLOAT(.5), MAKE_FLOAT(.5));
    ElVisFloat3 secondLevelAmbiguous = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(.5), MAKE_FLOAT(.5));



    if( MatchVisual3Contours )
    {
        bool oneIsovalueIsValid = false;
        int numIsovalues = Isovalues.size();
        ELVIS_PRINTF("CategorizeContourPixels: Num isovalues: %d\n", numIsovalues);
        for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
        {
            ElVisFloat isovalue = Isovalues[isoValueIndex];
            ELVIS_PRINTF("CategorizeContourPixels: testing isovalue: %f\n", isovalue);
            bool lowerThanOneValue = (isovalue <= c0) || (isovalue <= c1) || (isovalue <= c2) || (isovalue <= c3);
            bool higherThanOneValue = (isovalue >= c0) || (isovalue >= c1) || (isovalue >= c2) || (isovalue >= c3);

            if( lowerThanOneValue && higherThanOneValue )
            {
                oneIsovalueIsValid = true;
                break;

            }
        }

        if( !oneIsovalueIsValid )
        {
            raw_color_buffer[launch_index] = visual3BackgroundColor;
            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
            SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
        }
    }
    else
    {
        int numIsovalues = Isovalues.size();
        ELVIS_PRINTF("CategorizeContourPixels: Num isovalues: %d\n", numIsovalues);
        for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
        {
            ElVisFloat isovalue = Isovalues[isoValueIndex];
            ELVIS_PRINTF("CategorizeContourPixels: testing isovalue: %f\n", isovalue);
            bool lowerThanOneValue = (isovalue <= c0) || (isovalue <= c1) || (isovalue <= c2) || (isovalue <= c3);
            bool higherThanOneValue = (isovalue >= c0) || (isovalue >= c1) || (isovalue >= c2) || (isovalue >= c3);

            if( lowerThanOneValue && higherThanOneValue )
            {
                ELVIS_PRINTF("(%d, %d), Isovalue %f and corners %f, %f, %f, %f\n", launch_index.x, launch_index.y, isovalue, c0, c1, c2, c3);
                // BLACK - contour
                raw_color_buffer[launch_index] = contourColor;
                color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
                normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));

                // Clear out the sample buffer to prevent the color mapper from kicking in.
                SampleBuffer[launch_index] = ELVIS_FLOAT_MAX;
                //oneIsovalueIsValid = true;
            }
        }
    }

    return;

    //if( !oneIsovalueIsValid )
    //{
    //    // If the element types are all different, then I can't use interval arithmetic.
    //    if( ElementIdAtIntersectionBuffer[c0_index] != ElementIdAtIntersectionBuffer[c1_index] ||
    //        ElementIdAtIntersectionBuffer[c0_index] != ElementIdAtIntersectionBuffer[c2_index] ||
    //        ElementIdAtIntersectionBuffer[c0_index] != ElementIdAtIntersectionBuffer[c3_index] ||
    //        ElementTypeAtIntersectionBuffer[c0_index] != ElementTypeAtIntersectionBuffer[c1_index] || 
    //        ElementTypeAtIntersectionBuffer[c0_index] != ElementTypeAtIntersectionBuffer[c2_index] ||
    //        ElementTypeAtIntersectionBuffer[c0_index] != ElementTypeAtIntersectionBuffer[c3_index])
    //    {            
    //        // Mark the element boundary.
    //        raw_color_buffer[launch_index] = elementBoundaryColor;
    //        color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
    //        normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    //        return;
    //    }
    //    if( ElementTypeAtIntersectionBuffer[c0_index] == 0 )
    //    {
    //        // Hex
    //        // Vertical
    //        // White for NO
    //        raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
    //        color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
    //        normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    //    }
    //    else
    //    {
    //        // No Subdivisions
    //        ElVis::Interval<ElVisFloat> edge0 = EvaluatePrismEdge(ElementIdAtIntersectionBuffer[c0_index], c0_index, c1_index);
    //        ElVis::Interval<ElVisFloat> edge1 = EvaluatePrismEdge(ElementIdAtIntersectionBuffer[c0_index], c0_index, c2_index);
    //        ElVis::Interval<ElVisFloat> edge2 = EvaluatePrismEdge(ElementIdAtIntersectionBuffer[c0_index], c1_index, c3_index);
    //        ElVis::Interval<ElVisFloat> edge3 = EvaluatePrismEdge(ElementIdAtIntersectionBuffer[c0_index], c2_index, c3_index);

    //        bool mayContainAnIsovalue = false;
    //        for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
    //        {       
    //            float isovalue = Isovalues[isoValueIndex];
    //            if(edge0.Contains(isovalue) || edge1.Contains(isovalue) || 
    //                edge2.Contains(isovalue) || edge3.Contains(isovalue) )
    //            {
    //                mayContainAnIsovalue = true;
    //            }
    //        }

    //        if( !mayContainAnIsovalue )
    //        {
    //            // Definitely not: WHITE
    //            raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
    //            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
    //            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    //            return;
    //        }

    //        // Still not quite right, but at least the timings will be on the correct order now.

    //        // 1 Subdivision
    //        //edge0 = SubdivideInterval1(ElementIdAtIntersectionBuffer[c0_index], 
    //        //        ReferencePointAtIntersectionBuffer[c0_index],
    //        //        ReferencePointAtIntersectionBuffer[c2_index], 
    //        //        make_float2(-1.0, -1.0), make_float2(-1.0, 1.0));
    //        //edge1 = SubdivideInterval1(ElementIdAtIntersectionBuffer[c0_index], 
    //        //    ReferencePointAtIntersectionBuffer[c1_index],
    //        //    ReferencePointAtIntersectionBuffer[c3_index], 
    //        //    make_float2(1.0, -1.0), make_float2(1.0, 1.0));

    //        //edge2 = SubdivideInterval1(ElementIdAtIntersectionBuffer[c0_index], 
    //        //    ReferencePointAtIntersectionBuffer[c0_index],
    //        //    ReferencePointAtIntersectionBuffer[c1_index], 
    //        //    make_float2(-1.0, -1.0), make_float2(1.0, -1.0));

    //        //edge3 = SubdivideInterval1(ElementIdAtIntersectionBuffer[c0_index], 
    //        //    ReferencePointAtIntersectionBuffer[c2_index],
    //        //    ReferencePointAtIntersectionBuffer[c3_index], 
    //        //    make_float2(-1.0, 1.0), make_float2(1.0, 1.0));


    //        
    //        // 2 Subdivisions
    //        //edge0 = SubdivideInterval2(ElementIdAtIntersectionBuffer[c0_index], 
    //        //        ReferencePointAtIntersectionBuffer[c0_index],
    //        //        ReferencePointAtIntersectionBuffer[c2_index], 
    //        //        make_float2(-1.0, -1.0), make_float2(-1.0, 1.0));
    //        //edge1 = SubdivideInterval2(ElementIdAtIntersectionBuffer[c0_index], 
    //        //    ReferencePointAtIntersectionBuffer[c1_index],
    //        //    ReferencePointAtIntersectionBuffer[c3_index], 
    //        //    make_float2(1.0, -1.0), make_float2(1.0, 1.0));

    //        //edge2 = SubdivideInterval2(ElementIdAtIntersectionBuffer[c0_index], 
    //        //    ReferencePointAtIntersectionBuffer[c0_index],
    //        //    ReferencePointAtIntersectionBuffer[c1_index], 
    //        //    make_float2(-1.0, -1.0), make_float2(1.0, -1.0));

    //        //edge3 = SubdivideInterval2(ElementIdAtIntersectionBuffer[c0_index], 
    //        //    ReferencePointAtIntersectionBuffer[c2_index],
    //        //    ReferencePointAtIntersectionBuffer[c3_index], 
    //        //    make_float2(-1.0, 1.0), make_float2(1.0, 1.0));

    //        // 3 Subdivisions
    //        //edge0 = SubdivideInterval3(ElementIdAtIntersectionBuffer[c0_index], 
    //        //        ReferencePointAtIntersectionBuffer[c0_index],
    //        //        ReferencePointAtIntersectionBuffer[c2_index], 
    //        //        make_float2(-1.0, -1.0), make_float2(-1.0, 1.0));
    //        //edge1 = SubdivideInterval3(ElementIdAtIntersectionBuffer[c0_index], 
    //        //    ReferencePointAtIntersectionBuffer[c1_index],
    //        //    ReferencePointAtIntersectionBuffer[c3_index], 
    //        //    make_float2(1.0, -1.0), make_float2(1.0, 1.0));

    //        //edge2 = SubdivideInterval3(ElementIdAtIntersectionBuffer[c0_index], 
    //        //    ReferencePointAtIntersectionBuffer[c0_index],
    //        //    ReferencePointAtIntersectionBuffer[c1_index], 
    //        //    make_float2(-1.0, -1.0), make_float2(1.0, -1.0));

    //        //edge3 = SubdivideInterval3(ElementIdAtIntersectionBuffer[c0_index], 
    //        //    ReferencePointAtIntersectionBuffer[c2_index],
    //        //    ReferencePointAtIntersectionBuffer[c3_index], 
    //        //    make_float2(-1.0, 1.0), make_float2(1.0, 1.0));

    //        //if( launch_index.x == 431 && launch_index.y == 166 )
    //        //{
    //        //    ELVIS_PRINTF("Original Edge (%f, %f), subdivided edge (%f, %f)\n",
    //        //        edge1.GetLow(), edge1.GetHigh(), subdividedInterval0.GetLow(), subdividedInterval0.GetHigh());

    //        //    ELVIS_PRINTF("Original Edge (%f, %f), subdivided edge (%f, %f)\n",
    //        //        edge2.GetLow(), edge2.GetHigh(), subdividedInterval1.GetLow(), subdividedInterval1.GetHigh());

    //        //}

    //        mayContainAnIsovalue = false;
    //        for(int isoValueIndex = 0; isoValueIndex < Isovalues.size(); ++isoValueIndex)
    //        {       
    //            float isovalue = Isovalues[isoValueIndex];
    //            if(edge0.Contains(isovalue) || edge1.Contains(isovalue) || 
    //                edge2.Contains(isovalue) || edge3.Contains(isovalue) )
    //            {
    //                mayContainAnIsovalue = true;
    //            }
    //        }

    //        if( mayContainAnIsovalue )
    //        {
    //            raw_color_buffer[launch_index] = ambiguousColor;
    //            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
    //            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    //        }
    //        else
    //        {
    //            // Definitely not: WHITE
    //            raw_color_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(1.0), MAKE_FLOAT(1.0), MAKE_FLOAT(1.0));
    //            color_buffer[launch_index] = ConvertToColor(raw_color_buffer[launch_index]);
    //            normal_buffer[launch_index] = MakeFloat3(MAKE_FLOAT(0.0), MAKE_FLOAT(0.0), MAKE_FLOAT(0.0));
    //        }
    //        
    //    }
    //}
}

#endif
