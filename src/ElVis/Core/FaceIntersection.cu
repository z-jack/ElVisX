#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
// The MIT License
//
// Copyright (c) 2006 Scientific Computing and Imaging Institute,
// University of Utah (USA)
//
// License for the specific language governing rights and limitations under
// Permission is hereby granted, free of charge, to any person obtaining a
// copy of this software and associated documentation files (the "Software"),
// to deal in the Software without restriction, including without limitation
// the rights to use, copy, modify, merge, publish, distribute, sublicense,
// and/or sell copies of the Software, and to permit persons to whom the
// Software is furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included
// in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
// OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
// THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
// DEALINGS IN THE SOFTWARE.
//
///////////////////////////////////////////////////////////////////////////////

#ifndef ELVIS_CORE_FACE_INTERSECTION_CU
#define ELVIS_CORE_FACE_INTERSECTION_CU

__device__ VolumeRenderingPayload FindNextFaceIntersection(const ElVisFloat3& origin, 
           const ElVisFloat3& rayDirection)
{
  VolumeRenderingPayload payload;
  payload.Initialize();

  optix::Ray ray = optix::make_Ray(ConvertToFloat3(origin), ConvertToFloat3(rayDirection), 2, 1e-3, RT_DEFAULT_MAX);

  // do linear faces first, since they are fast.  Intersections with linear 
  // faces may help weed out bad curved matches.
  rtTrace(PlanarFaceGroup, ray, payload);
  rtTrace(CurvedFaceGroup, ray, payload);

  return payload;
}

__device__ void FaceBoundingBox(int globalFaceIdx, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;

    ElVisFloat3 p0 = FaceInfoBuffer[globalFaceIdx].MinExtent;
    ElVisFloat3 p1 = FaceInfoBuffer[globalFaceIdx].MaxExtent;

    //rtPrintf("FaceBoundingBoxProgram: (%f, %f, %f) - (%f, %f, %f)\n", 
    //  p0.x, p0.y, p0.z, p1.x, p1.y, p1.z);
    aabb->m_min = make_float3(p0.x, p0.y, p0.z);
    aabb->m_max = make_float3(p1.x, p1.y, p1.z);
}

RT_PROGRAM void PlanarFaceBoundingBoxProgram(int primitiveId, float result[6])
{
  FaceBoundingBox(PlanarFaceToGlobalIdxMap[primitiveId], result);
}

RT_PROGRAM void CurvedFaceBoundingBoxProgram(int primitiveId, float result[6])
{
  FaceBoundingBox(CurvedFaceToGlobalIdxMap[primitiveId], result);
}

#endif
